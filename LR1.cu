#include "hip/hip_runtime.h"
%%cu

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void raise_power(int two, int power, int *res)
{
  int i, idx = blockDim.x * blockIdx.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (i = idx; i < power; i += offset)
  {
	res[i] = pow(two, i);
  }
}

int main()
{
  int res;
  int i;
  int two;
  int power;

  if(hipSetDevice(0)!= hipSuccess)
  {
    cerr << "ERROR: CUDA validation failed.\n";
	return 0;
  }
  
  two = 2;
  power = 32;
 
  int* arr = (int*)malloc(sizeof(int) * power);
 
  int* dev_arr;

  hipError_t err = hipMalloc(&dev_arr, sizeof(int) * power);
 
  if (err != hipSuccess) 
  {
	cerr << "ERROR: bad Malloc\n";
	return 0;
  }

  if (hipMemcpy(dev_arr, arr, sizeof(int) * power, hipMemcpyHostToDevice) != hipSuccess) 
  {
	cerr << "ERROR: bad Memcpy\n";
	return 0;
  }
 
  raise_power <<<256, 256>>> (two, power, dev_arr);
 

  if (hipGetLastError() != hipSuccess) 
  {
	cerr << "ERROR: kernel error\n";
  }

  if (hipMemcpy(arr, dev_arr, sizeof(int) * power, hipMemcpyDeviceToHost) != hipSuccess) 
  {
	cerr << "ERROR: bad Memcpy\n";
	return 0;
  }

  if (hipFree(dev_arr) != hipSuccess) 
  {
	cerr << "ERROR: bad Free\n";
	return 0;
  }
 
  for (i = 0; i < power; i++)
  {
	printf("%d\n", arr[i]);
  }
  free(arr);
  return 0;
}



